#include "hip/hip_runtime.h"
#ifndef assert
#define assert(e)  \
    if (!(e)) { \
        printf("failed assertion `%s'\n", #e); \
        THError("aborting..."); \
    };
#endif

#include "utils.h"

static texture<float, hipTextureType2DLayered> texRef2;

__global__ void extractInterpolateBDHWKernel(float* outptr, int outstr0, int outstr1, int outstr2, int outstr3, int outx, int outy, float y1, float x1, float y2, float x2, float y3, float x3, float y4, float x4, int inLayers)
{

   const int pixIdxX = blockIdx.x*blockDim.x+threadIdx.x;
   const int pixIdxY = blockIdx.y*blockDim.y+threadIdx.y;

   const float coordx0 = (float)(pixIdxX)/outx;
   const float coordy0 = (float)(pixIdxY)/outy;

   // we put some offset (y_i, x_i) are the input coordinates of the output corners (1 : top-left, 2 : top-right, 3 : bot-right, 4 : bot-left)
   const float upinter = (x1+(coordx0*(x2-x1)));
   const float downinter = (x4+(coordx0*(x3-x4)));
   const float leftinter = (y1+(coordy0*(y4-y1)));
   const float rightinter = (y2+(coordy0*(y3-y2)));

   const float coordx = upinter + coordy0*(downinter - upinter);
   const float coordy = leftinter + coordx0*(rightinter - leftinter);

   int i;

//   if (coordx<1 && coordy<1 && coordx0 <1 && coordy0 <1)
   if(pixIdxX<outx && pixIdxY<outy)
   {
   // read :
      for (i=0; i<inLayers; i++)
      {
          outptr[outstr1*i + outstr2*pixIdxY + outstr3*pixIdxX] = tex2DLayered(texRef2, coordx, coordy, i);
      }
   }
}

static int texfuncs_ExtractInterpolate_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  hipArray* imgarray = (hipArray *) lua_touserdata(L, 3);

  int outy = luaT_getfieldcheckint(L, 1, "targety");
  int outx = luaT_getfieldcheckint(L, 1, "targetx");
  int y1int = luaT_getfieldcheckint(L, 1, "y1");
  int y2int = luaT_getfieldcheckint(L, 1, "y2");
  int y3int = luaT_getfieldcheckint(L, 1, "y3");
  int y4int = luaT_getfieldcheckint(L, 1, "y4");
  int x1int = luaT_getfieldcheckint(L, 1, "x1");
  int x2int = luaT_getfieldcheckint(L, 1, "x2");
  int x3int = luaT_getfieldcheckint(L, 1, "x3");
  int x4int = luaT_getfieldcheckint(L, 1, "x4");

  input = THCudaTensor_newContiguous(state, input); // should be contiguous already
  
  int bs       = input->size[0];
  int nPlanes  = input->size[1];
  int ih       = input->size[2];
  int iw       = input->size[3];
//  assert(nPlanes==3);

  THCudaTensor_resize4d(state, output, bs, nPlanes,  outy, outx);
  
  float y1 = ((float)y1int-1)/(float)(ih-1);
  float y2 = ((float)y2int-1)/(float)(ih-1);
  float y3 = ((float)y3int-1)/(float)(ih-1);
  float y4 = ((float)y4int-1)/(float)(ih-1);
  float x1 = ((float)x1int-1)/(float)(iw-1);
  float x2 = ((float)x2int-1)/(float)(iw-1);
  float x3 = ((float)x3int-1)/(float)(iw-1);
  float x4 = ((float)x4int-1)/(float)(iw-1);
  
  
  hipError_t result;
  hipError_t err;

  float * outptr=THCudaTensor_data(state, output);
  
    texRef2.addressMode[0]   = hipAddressModeBorder;
    texRef2.addressMode[1]   = hipAddressModeBorder;
    texRef2.filterMode       = hipFilterModeLinear;
    texRef2.normalized       = 1;
	
    hipBindTextureToArray(texRef2, imgarray);


    int outstr0    = output->stride[0];
    int outstr1    = output->stride[1];
    int outstr2    = output->stride[2];
    int outstr3    = output->stride[3];
    
    dim3 blockstiled((outx+7)/8, (outy+3)/4, bs);
    dim3 threadstiled(8,4);

//    dim3 blocks((outx+31)/32, outy);
//    dim3 threads(32);
    dim3 blocks((outx+7)/8, (outy+7)/8);
    dim3 threads(8,8);


    //printf("%f, %f, %f, %f, %f, %f, %f, %f\n", y1, x1, y2, x2, y3, x3, y4, x4);
    
    extractInterpolateBDHWKernel <<<blocks, threads>>>(outptr, outstr0, outstr1, outstr2, outstr3, outx, outy, y1, x1, y2, x2, y3, x3, y4, x4, bs*nPlanes);

   err = hipGetLastError();
 
    hipUnbindTexture(texRef2);

  // check for errors
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in ExtractInterpolate.updateOutput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
 
  // final cut:
  THCudaTensor_free(state, input); 
  //THCudaTensor_free(tmp); 
  //THCudaTensor_select(output, NULL, dimension, 0);

  return 1;
}



static int texfuncs_ExtractInterpolate_initCudaArray(lua_State *L)
{
    THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray* imgarray;

    int bs       = input->size[0];
    int nPlanes  = input->size[1];
    int ih       = input->size[2];
    int iw       = input->size[3];
//    assert(nPlanes==3);
 
    hipExtent ex = make_hipExtent(iw, ih, bs*nPlanes);

    hipError_t result;

    result = hipMalloc3DArray(&imgarray, &channelDesc, ex, hipArrayLayered);
    if (result != hipSuccess) {
        fprintf(stderr, "hipMalloc3DArray -  %s\n", hipGetErrorString(result));
        return 1;
    }  

    lua_pushlightuserdata (L, imgarray);
    return 1;
}


static int texfuncs_ExtractInterpolate_copyIntoArray(lua_State *L)
{
    THCState *state = getCutorchState(L);
    THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
    hipArray* imgarray = (hipArray *) lua_touserdata(L, 3);

    int bs       = input->size[0];
    int nPlanes  = input->size[1];
    int ih       = input->size[2];
    int iw       = input->size[3];
//    assert(nPlanes==3);

    hipError_t result;

    hipMemcpy3DParms myParms = {0};
    memset(&myParms, 0, sizeof(myParms));
    myParms.srcPtr.pitch = sizeof(float) * iw;
//    myParms.srcPtr.ptr = THCudaTensor_data(state, input);
    myParms.srcPtr.ptr = THCudaTensor_data(state, input);
    myParms.srcPtr.xsize = iw;
    myParms.srcPtr.ysize = ih;

    myParms.srcPos.x = 0;
    myParms.srcPos.y = 0;
    myParms.srcPos.z = 0;
  
    myParms.dstArray = imgarray;

    myParms.dstPos.x = 0;
    myParms.dstPos.y = 0;
    myParms.dstPos.z = 0;

    myParms.extent.width = iw;
    myParms.extent.depth = bs*nPlanes;
    myParms.extent.height = ih;

    myParms.kind = hipMemcpyDeviceToDevice;

    result = hipMemcpy3DAsync(&myParms);
    if (result != hipSuccess) {
        fprintf(stderr, "hipMemcpy3D - failed to copy 1 - %s\n", hipGetErrorString(result));
        return 1;
    }
    return 1;
}


static int texfuncs_ExtractInterpolate_destroyArray(lua_State *L)
{
    hipArray* imgarray = (hipArray *) lua_touserdata(L, 2);
    hipError_t result;
    result = hipFreeArray(imgarray);
    if (result != hipSuccess) {
        fprintf(stderr, "hipFreeArray - %s\n", hipGetErrorString(result));
        return 1;
    }
    return 0;
}





static int texfuncs_ExtractInterpolate_updateGradInput(lua_State *L)
{
  return 1;
}

static const struct luaL_Reg texfuncs_ExtractInterpolate__ [] = {
  {"texfuncs_ExtractInterpolate_updateOutput", texfuncs_ExtractInterpolate_updateOutput},
  {"texfuncs_ExtractInterpolate_initCudaArray", texfuncs_ExtractInterpolate_initCudaArray},
  {"texfuncs_ExtractInterpolate_copyIntoArray", texfuncs_ExtractInterpolate_copyIntoArray},
  {"texfuncs_ExtractInterpolate_destroyArray", texfuncs_ExtractInterpolate_destroyArray},
  {"", },
  {NULL, NULL}
};

static void texfuncs_ExtractInterpolate_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, texfuncs_ExtractInterpolate__, "nn");
  lua_pop(L,1);
}
